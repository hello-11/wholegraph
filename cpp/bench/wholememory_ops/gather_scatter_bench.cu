#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <unistd.h>
#include <getopt.h>
#include <sys/time.h>

#include <iostream>
#include <string>
#include <string_view>

#include <wholememory/tensor_description.h>
#include <wholememory/wholememory.h>
#include <wholememory/wholememory_op.h>

#include "../common/wholegraph_benchmark.hpp"
#include "parallel_utils.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/env_func_ptrs.hpp"
#include "wholememory/initialize.hpp"

#include "../../tests/wholememory/wholememory_test_utils.hpp"
#include "../../tests/wholememory_ops/embedding_test_utils.hpp"

namespace wholegraph::bench::gather_scatter{

typedef struct GatherScatterBenchParam {
  wholememory_matrix_description_t get_embedding_desc() const
  { 
    int64_t embedding_entry_count = get_embedding_entry_count();
    int64_t matrix_sizes[2] = {embedding_entry_count, embedding_dim};
    return wholememory_create_matrix_desc(
      matrix_sizes, embedding_stride, embedding_storage_offset, embedding_type);
  }
  wholememory_array_description_t get_indices_desc() const
  {
    int64_t indices_count = get_indices_count();
    return wholememory_create_array_desc(indices_count, indices_storage_offset, indices_type);
  }
  wholememory_matrix_description_t get_output_desc() const
  {
    int64_t indices_count = get_indices_count();
    int64_t output_sizes[2] = {indices_count, embedding_dim};
    return wholememory_create_matrix_desc(
      output_sizes, output_stride, output_storage_offset, output_type);
  }

  int64_t get_embedding_granularity() const
  {
    return embedding_stride * wholememory_dtype_get_element_size(embedding_type);
  }

  int64_t get_embedding_table_size() const {
    return embedding_table_size;
  }
  int64_t get_gather_size() const {
    return gather_size;
  }

  wholememory_memory_type_t get_memory_type() const {
    return memory_type;
  }

  wholememory_memory_location_t get_memory_location() const {
    return memory_location;
  }
  int get_loop_count () const {
    return loop_count;
  }
  std::string get_test_type() const {
    return test_type;
  }

  int64_t get_embedding_dim() const {
    return embedding_dim;
  }
  wholememory_dtype_t get_embedding_type() const {
    return embedding_type;
  }

  GatherScatterBenchParam& set_memory_type(wholememory_memory_type_t new_memory_type)
  {
    memory_type = new_memory_type;
    return *this;
  }
  GatherScatterBenchParam& set_memory_location(wholememory_memory_location_t new_memory_location)
  {
    memory_location = new_memory_location;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_table_size(int64_t new_embedding_table_size) {
    int64_t entry_size =  wholememory_dtype_get_element_size(embedding_type) * get_embedding_dim();
    embedding_table_size = (new_embedding_table_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  GatherScatterBenchParam& set_gather_size(int64_t new_gather_size) {
    int64_t entry_size =  wholememory_dtype_get_element_size(embedding_type) * get_embedding_dim();
    gather_size = (new_gather_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_dim(int64_t new_embedding_dim) {
    embedding_dim = new_embedding_dim;
    if (embedding_stride != embedding_dim) embedding_stride = embedding_dim;
    if (output_stride != embedding_dim) output_stride = embedding_dim;
    int64_t entry_size = wholememory_dtype_get_element_size(embedding_type) * embedding_dim;
    embedding_table_size = (embedding_table_size + entry_size - 1) / entry_size * entry_size;
    gather_size = (gather_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  
  GatherScatterBenchParam& set_loop_count(int new_loop_count) {
    loop_count = new_loop_count;
    return *this;
  }
  
  GatherScatterBenchParam& set_test_type(std::string new_test_type) {
    test_type = new_test_type;
    return *this;
  }


  private: 
  int64_t get_embedding_entry_count() const {
    return embedding_table_size / wholememory_dtype_get_element_size(embedding_type) / embedding_dim;
  }
  int64_t get_indices_count() const {
    return gather_size / wholememory_dtype_get_element_size(embedding_type)/ embedding_dim;
  }

  GatherScatterBenchParam& set_embedding_stride(int64_t new_embedding_stride)
  {
    embedding_stride = new_embedding_stride;
    return *this;
  }
  GatherScatterBenchParam& set_output_stride(int64_t new_output_stride)
  {
    output_stride = new_output_stride;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_type(wholememory_dtype_t new_embedding_type)
  {
    embedding_type = new_embedding_type;
    return *this;
  }
  GatherScatterBenchParam& set_indices_type(wholememory_dtype_t new_indices_type)
  {
    indices_type = new_indices_type;
    return *this;
  }
  GatherScatterBenchParam& set_output_type(wholememory_dtype_t new_output_type)
  {
    output_type = new_output_type;
    return *this;
  }
  wholememory_memory_type_t memory_type         = WHOLEMEMORY_MT_CHUNKED;
  wholememory_memory_location_t memory_location = WHOLEMEMORY_ML_DEVICE;
  int64_t embedding_table_size = 1024000LL;
  int64_t gather_size = 1024; 
  int64_t embedding_dim                         = 32;
  int loop_count = 20;
  std::string test_type = "gather"; //gather or scatter

  int64_t embedding_stride                      = 32;
  int64_t output_stride                         = 32;
  wholememory_dtype_t embedding_type            = WHOLEMEMORY_DT_FLOAT;
  wholememory_dtype_t indices_type              = WHOLEMEMORY_DT_INT64;
  wholememory_dtype_t output_type               = WHOLEMEMORY_DT_FLOAT;
  int64_t embedding_storage_offset              = 0;
  int64_t indices_storage_offset                = 0;
  int64_t output_storage_offset                 = 0;
} GatherScatterBenchParam;

std::string get_memory_type_string(wholememory_memory_type_t memory_type) {
  std::string str; 
  switch (memory_type)
  {
  case WHOLEMEMORY_MT_NONE:
    str = "WHOLEMEMORY_MT_NONE";
    break;
  case WHOLEMEMORY_MT_CONTINUOUS:
    str = "WHOLEMEMORY_MT_CONTINUOUS";
    break;
  case WHOLEMEMORY_MT_CHUNKED:
    str = "WHOLEMEMORY_MT_CHUNKED";
    break;
  case WHOLEMEMORY_MT_DISTRIBUTED:
    str = "WHOLEMEMORY_MT_DISTRIBUTED";
    break;
  default:
    break;
  }
  return str;
}

std::string get_memory_location_string(wholememory_memory_location_t memory_location) {
  std::string str;
  switch (memory_location)
  {
  case WHOLEMEMORY_ML_NONE:
    str = "WHOLEMEMORY_ML_NONE";
    break;
  case WHOLEMEMORY_ML_DEVICE:
    str =  "WHOLEMEMORY_ML_DEVICE";
    break;
  case WHOLEMEMORY_ML_HOST:
    str = "WHOLEMEMORY_ML_HOST";
    break;
  default:
    break;
  }
  return str;
}

void gather_scatter_benchmark(GatherScatterBenchParam &params) {
  int g_dev_count = ForkGetDeviceCount(); 
  EXPECT_GE(g_dev_count, 1);
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, g_dev_count);
  MultiProcessRun(
    g_dev_count,
    [&params, &pipes](int world_rank, int world_size) {
      wholememory_init(0);

      hipSetDevice(world_rank);

      wholememory_comm_t wm_comm = create_communicator_by_pipes(pipes, world_rank, world_size);

      auto embedding_desc = params.get_embedding_desc();
      auto indices_desc = params.get_indices_desc();
      auto output_desc = params.get_output_desc();
      std::string test_type = params.get_test_type();
      size_t embedding_entry_size = params.get_embedding_granularity();
      wholememory_handle_t embedding_handle;
      wholememory_malloc(&embedding_handle,
                                   params.get_embedding_table_size(),
                                   wm_comm,
                                   params.get_memory_type(),
                                   params.get_memory_location(),
                                   embedding_entry_size);

      hipStream_t stream;
      hipStreamCreate(&stream);

      void *dev_indices = nullptr, *dev_gather_buffer = nullptr;
      void *host_indices = nullptr;
      size_t gather_buffer_size  = params.get_gather_size();
      size_t indices_buffer_size = wholememory_get_memory_size_from_array(&indices_desc);

      hipHostMalloc(&host_indices, indices_buffer_size);
      hipMalloc(&dev_indices, indices_buffer_size);
      hipMalloc(&dev_gather_buffer, gather_buffer_size);

      wholememory_ops::testing::device_random_init_local_embedding_table(
        embedding_handle, embedding_desc, stream);
      wholememory_ops::testing::host_random_init_indices(
        host_indices, indices_desc, embedding_desc.sizes[0]);
      hipMemcpyAsync(dev_indices,
                                host_indices,
                                wholememory_get_memory_size_from_array(&indices_desc),
                                hipMemcpyHostToDevice,
                                stream);
      hipStreamSynchronize(stream);
      wholememory_communicator_barrier(wm_comm);

      wholememory_tensor_t embedding_tensor;
      wholememory_tensor_description_t embedding_tensor_desc;
      wholememory_copy_matrix_desc_to_tensor(&embedding_tensor_desc, &embedding_desc);
      wholememory_make_tensor_from_handle(
                  &embedding_tensor, embedding_handle, &embedding_tensor_desc);

      wholememory_tensor_t indices_tensor, output_tensor;
      wholememory_tensor_description_t indices_tensor_desc, output_tensor_desc;
      wholememory_copy_array_desc_to_tensor(&indices_tensor_desc, &indices_desc);
      wholememory_copy_matrix_desc_to_tensor(&output_tensor_desc, &output_desc);
      wholememory_make_tensor_from_pointer(&indices_tensor, dev_indices, &indices_tensor_desc);
      wholememory_make_tensor_from_pointer(
                  &output_tensor, dev_gather_buffer, &output_tensor_desc);
      hipStreamSynchronize(stream);
      wholememory_communicator_barrier(wm_comm);

      for (int i = 0; i < 10; i++) {
        if (test_type.compare("gather") == 0) {
          wholememory_gather(embedding_tensor,
                             indices_tensor,
                             output_tensor,
                             wholememory::get_default_env_func(),
                             stream);
        }
        else if (test_type.compare("scatter") == 0) {
          wholememory_scatter(output_tensor,
                              indices_tensor,
                              embedding_tensor,
                              wholememory::get_default_env_func(),
                              stream);
        }
      }
      hipStreamSynchronize(stream);
      hipDeviceSynchronize();
      wholememory_communicator_barrier(wm_comm);

      int loop_count = params.get_loop_count();
      struct timeval tv_s, tv_e;
      if (test_type.compare("gather") == 0) {
        gettimeofday(&tv_s, nullptr);
        for (int i = 0; i < loop_count; i++) {
          wholememory_gather(embedding_tensor,
                            indices_tensor,
                            output_tensor,
                            wholememory::get_default_env_func(),
                            stream);
        }
        hipStreamSynchronize(stream);
        gettimeofday(&tv_e, nullptr);
      }
      else if (test_type.compare("scatter") == 0) {
          gettimeofday(&tv_s, nullptr);
          for (int i = 0; i < loop_count; i++) {
          wholememory_scatter(output_tensor,
                                      indices_tensor,
                                      embedding_tensor,
                                      wholememory::get_default_env_func(),
                                      stream);
          }
          hipStreamSynchronize(stream);
          gettimeofday(&tv_e, nullptr);
      }
      else {
        throw std::invalid_argument("Invalid test type");
      }
      int time_us = TIME_DIFF_US(tv_s, tv_e);
      double bw = gather_buffer_size / time_us / 1e3 * loop_count;
      wholememory_communicator_barrier(wm_comm);

      std::vector<double> recv_vec(world_size);
      wm_comm->host_allgather(&bw, recv_vec.data(), 1, WHOLEMEMORY_DT_DOUBLE);
      
      double min_bw, max_bw, avg_bw;
      min_bw = max_bw = recv_vec[0];
      avg_bw = 0.0;
      for (int i = 0; i < world_size; i++) {
          min_bw = std::min(min_bw, recv_vec[i]);
          max_bw = std::max(max_bw, recv_vec[i]);
          avg_bw += recv_vec[i];
      }
      avg_bw /= world_size;
      double emb_size_mb = (double)params.get_embedding_table_size()/1024.0/1024.0;
      double gather_size_mb = (double)params.get_gather_size()/1024.0/1024.0;
      if (world_rank == 0) {
        printf("%s, world_size=%d, memoryType=%s, memoryLocation=%s, elt_size=%ld, embeddingDim=%ld, embeddingTableSize=%.2lf MB, gatherSize=%.2lf MB, minBW=%.2lf GB/s, maxBW=%.2lf GB/s, "
            "avgBW=%.2lf GB/s\n",
            test_type.c_str(), world_size, get_memory_type_string(params.get_memory_type()).c_str(), get_memory_location_string(params.get_memory_location()).c_str(), wholememory_dtype_get_element_size(params.get_embedding_type()), params.get_embedding_dim(), emb_size_mb, gather_size_mb, min_bw, max_bw,
            avg_bw);
      }

      wholememory_destroy_tensor(indices_tensor);
      wholememory_destroy_tensor(output_tensor);

      hipHostFree(host_indices);
      hipFree(dev_indices);
      hipFree(dev_gather_buffer);

      wholememory_destroy_tensor(embedding_tensor);

      wholememory_free(embedding_handle);

      wholememory::destroy_all_communicators();

      wholememory_finalize();
    },
    true);
}

}  // namespace wholegraph::bench::gather_catter

int main(int argc, char** argv) {
    wholegraph::bench::gather_scatter::GatherScatterBenchParam params;
    const char* optstr = "ht:l:e:g:d:c:f:";
    struct option opts[] = {
        {"help", no_argument, NULL, 'h'},
        {"memory_type", required_argument, NULL, 't'},        // 0: None, 1: Continuous, 2: Chunked, 3 Distributed
        {"memory_location", required_argument, NULL, 'l'},    // 0: None, 1: Device, 2: Host
        {"embedding_table_size", required_argument, NULL, 'e'},
        {"gather_size", required_argument, NULL, 'g'},
        {"embedding_dim", required_argument, NULL, 'd'},
        {"loop_count", required_argument, NULL, 'c'},
        {"test_type", required_argument, NULL, 'f'}                   //test_type: gather or scatter
    };

    const char *usage = "Usage: %s [options]\n"
                  "Options:\n"
                  "  -h, --help      display this help and exit\n"
                  "  -t, --memory_type   specify wholememory type, 0: None, 1: Continuous, 2: Chunked, 3: Distributed\n"
                  "  -l, --memory_location    specify wholememory location, 0: None, 1: Device, 2: Host\n"
                  "  -e, --embedding_table_size    specify embedding table size\n"
                  "  -g, --gather_size    specify gather size\n"
                  "  -d, --embedding_dim    specify embedding dimension\n"
                  "  -c, --loop_count    specify loop count\n"
                  "  -f, --test_type    specify test type: gather or scatter\n";

    int c;
    bool has_option = false;
    while((c = getopt_long(argc, argv, optstr, opts, NULL)) != -1) {
        has_option = true;
        switch (c)
        {
        char *endptr;
        long val;
        case 'h':
            printf(usage, argv[0]);
            exit(EXIT_SUCCESS);
        case 't':
            val = strtol(optarg,&endptr, 10);
            if (*endptr != '\0' || val < 0 || val > 3) {
                printf("Invalid argument for option -t\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
              params.set_memory_type(static_cast<wholememory_memory_type_t>(val));
            break;
        case 'l':
            val = strtol(optarg,&endptr, 10);
            if (*endptr != '\0' || val < 0 || val > 2) {
                printf("Invalid argument for option -l\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            params.set_memory_location(static_cast<wholememory_memory_location_t>(val));
            break;
        case 'e':
            try {
                long long val = std::stoll(optarg);
                if (val < 0) {
                    throw std::invalid_argument("Negative value");
                }
                params.set_embedding_table_size(val);
            }
            catch (std::exception& e) {
                printf("Invalid argument for option -e\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        case 'g':
            try {
                long long val = std::stoll(optarg);
                if (val < 0) {
                    throw std::invalid_argument("Negative value");
                }
                params.set_gather_size(val);
            }
            catch (std::exception& e) {
                printf("Invalid argument for option -g\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        case 'd':
            try {
                int val = std::stoll(optarg);
                if (val < 0) {
                    throw std::invalid_argument("Negative value");
                }
                params.set_embedding_dim(val);
            }
            catch (std::exception& e) {
                printf("Invalid argument for option -d\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        case 'c':
            try {
                int val = std::stoi(optarg);
                if (val < 0) {
                    throw std::invalid_argument("Negative value");
                }
                params.set_loop_count(val);
            }
            catch (std::exception& e) {
                printf("Invalid argument for option -c\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        case 'f':
            if (strcmp(optarg, "gather") == 0) {
                params.set_test_type("gather");
            }
            else if (strcmp(optarg, "scatter") == 0) {
                params.set_test_type("scatter");
            }
            else {
                printf("Invalid argument for option -f\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        default:
            printf("Invalid or unrecognized option\n");
            printf(usage, argv[0]);
            exit(EXIT_FAILURE);
        }
    }
    if (!has_option) {
      printf("No option or argument is passed, use the default param\n");
    }
    wholegraph::bench::gather_scatter::gather_scatter_benchmark(params);
    return 0;
}