/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <unistd.h>
#include <getopt.h>
#include <sys/time.h>

#include <iostream>
#include <string>
#include <string_view>

#include <wholememory/tensor_description.h>
#include <wholememory/wholememory.h>
#include <wholememory/wholememory_op.h>

#include "../common/wholegraph_benchmark.hpp"
#include "parallel_utils.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/env_func_ptrs.hpp"
#include "wholememory/initialize.hpp"

#include "../../tests/wholememory/wholememory_test_utils.hpp"
namespace wholegraph::bench::gather_scatter{

typedef struct GatherScatterBenchParam {
  wholememory_matrix_description_t get_embedding_desc() const
  { 
    int64_t embedding_entry_count = get_embedding_entry_count();
    int64_t matrix_sizes[2] = {embedding_entry_count, embedding_dim};
    return wholememory_create_matrix_desc(
      matrix_sizes, embedding_stride, embedding_storage_offset, embedding_type);
  }
  wholememory_array_description_t get_indices_desc() const
  {
    int64_t indices_count = get_indices_count();
    return wholememory_create_array_desc(indices_count, indices_storage_offset, indices_type);
  }
  wholememory_matrix_description_t get_output_desc() const
  {
    int64_t indices_count = get_indices_count();
    int64_t output_sizes[2] = {indices_count, embedding_dim};
    return wholememory_create_matrix_desc(
      output_sizes, output_stride, output_storage_offset, output_type);
  }

  int64_t get_embedding_granularity() const
  {
    return embedding_stride * wholememory_dtype_get_element_size(embedding_type);
  }

  int64_t get_embedding_table_size() const {
    return embedding_table_size;
  }
  int64_t get_gather_size() const {
    return gather_size;
  }

  wholememory_memory_type_t get_memory_type() const {
    return memory_type;
  }

  wholememory_memory_location_t get_memory_location() const {
    return memory_location;
  }
  int get_loop_count () const {
    return loop_count;
  }
  std::string get_test_type() const {
    return test_type;
  }

  int64_t get_embedding_dim() const {
    return embedding_dim;
  }
  wholememory_dtype_t get_embedding_type() const {
    return embedding_type;
  }

  GatherScatterBenchParam& set_memory_type(wholememory_memory_type_t new_memory_type)
  {
    memory_type = new_memory_type;
    return *this;
  }
  GatherScatterBenchParam& set_memory_location(wholememory_memory_location_t new_memory_location)
  {
    memory_location = new_memory_location;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_table_size(int64_t new_embedding_table_size) {
    int64_t entry_size =  wholememory_dtype_get_element_size(embedding_type) * get_embedding_dim();
    embedding_table_size = (new_embedding_table_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  GatherScatterBenchParam& set_gather_size(int64_t new_gather_size) {
    int64_t entry_size =  wholememory_dtype_get_element_size(embedding_type) * get_embedding_dim();
    gather_size = (new_gather_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_dim(int64_t new_embedding_dim) {
    embedding_dim = new_embedding_dim;
    if (embedding_stride != embedding_dim) embedding_stride = embedding_dim;
    if (output_stride != embedding_dim) output_stride = embedding_dim;
    int64_t entry_size = wholememory_dtype_get_element_size(embedding_type) * embedding_dim;
    embedding_table_size = (embedding_table_size + entry_size - 1) / entry_size * entry_size;
    gather_size = (gather_size + entry_size - 1) / entry_size * entry_size;
    return *this;
  }
  
  GatherScatterBenchParam& set_loop_count(int new_loop_count) {
    loop_count = new_loop_count;
    return *this;
  }
  
  GatherScatterBenchParam& set_test_type(std::string new_test_type) {
    test_type = new_test_type;
    return *this;
  }


  private: 
  int64_t get_embedding_entry_count() const {
    return embedding_table_size / wholememory_dtype_get_element_size(embedding_type) / embedding_dim;
  }
  int64_t get_indices_count() const {
    return gather_size / wholememory_dtype_get_element_size(embedding_type)/ embedding_dim;
  }

  GatherScatterBenchParam& set_embedding_stride(int64_t new_embedding_stride)
  {
    embedding_stride = new_embedding_stride;
    return *this;
  }
  GatherScatterBenchParam& set_output_stride(int64_t new_output_stride)
  {
    output_stride = new_output_stride;
    return *this;
  }
  GatherScatterBenchParam& set_embedding_type(wholememory_dtype_t new_embedding_type)
  {
    embedding_type = new_embedding_type;
    return *this;
  }
  GatherScatterBenchParam& set_indices_type(wholememory_dtype_t new_indices_type)
  {
    indices_type = new_indices_type;
    return *this;
  }
  GatherScatterBenchParam& set_output_type(wholememory_dtype_t new_output_type)
  {
    output_type = new_output_type;
    return *this;
  }
  wholememory_memory_type_t memory_type         = WHOLEMEMORY_MT_CHUNKED;
  wholememory_memory_location_t memory_location = WHOLEMEMORY_ML_DEVICE;
  int64_t embedding_table_size = 1024000LL;
  int64_t gather_size = 1024; 
  int64_t embedding_dim                         = 32;
  int loop_count = 20;
  std::string test_type = "gather"; //gather or scatter

  int64_t embedding_stride                      = 32;
  int64_t output_stride                         = 32;
  wholememory_dtype_t embedding_type            = WHOLEMEMORY_DT_FLOAT;
  wholememory_dtype_t indices_type              = WHOLEMEMORY_DT_INT64;
  wholememory_dtype_t output_type               = WHOLEMEMORY_DT_FLOAT;
  int64_t embedding_storage_offset              = 0;
  int64_t indices_storage_offset                = 0;
  int64_t output_storage_offset                 = 0;
} GatherScatterBenchParam;

std::string get_memory_type_string(wholememory_memory_type_t memory_type) {
  std::string str; 
  switch (memory_type)
  {
  case WHOLEMEMORY_MT_NONE:
    str = "WHOLEMEMORY_MT_NONE";
    break;
  case WHOLEMEMORY_MT_CONTINUOUS:
    str = "WHOLEMEMORY_MT_CONTINUOUS";
    break;
  case WHOLEMEMORY_MT_CHUNKED:
    str = "WHOLEMEMORY_MT_CHUNKED";
    break;
  case WHOLEMEMORY_MT_DISTRIBUTED:
    str = "WHOLEMEMORY_MT_DISTRIBUTED";
    break;
  default:
    break;
  }
  return str;
}

std::string get_memory_location_string(wholememory_memory_location_t memory_location) {
  std::string str;
  switch (memory_location)
  {
  case WHOLEMEMORY_ML_NONE:
    str = "WHOLEMEMORY_ML_NONE";
    break;
  case WHOLEMEMORY_ML_DEVICE:
    str =  "WHOLEMEMORY_ML_DEVICE";
    break;
  case WHOLEMEMORY_ML_HOST:
    str = "WHOLEMEMORY_ML_HOST";
    break;
  default:
    break;
  }
  return str;
}

void gather_scatter_benchmark(GatherScatterBenchParam &params) {
  int g_dev_count = ForkGetDeviceCount(); 
  WHOLEMEMORY_CHECK_NOTHROW(g_dev_count >= 1);
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, g_dev_count);
  MultiProcessRun(
    g_dev_count,
    [&params, &pipes](int world_rank, int world_size) {
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_init(0) == WHOLEMEMORY_SUCCESS);

      WM_CUDA_CHECK_NO_THROW(hipSetDevice(world_rank));

      wholememory_comm_t wm_comm = create_communicator_by_pipes(pipes, world_rank, world_size);

      auto embedding_desc = params.get_embedding_desc();
      auto indices_desc = params.get_indices_desc();
      auto output_desc = params.get_output_desc();
      std::string test_type = params.get_test_type();
      size_t embedding_entry_size = params.get_embedding_granularity();
      wholememory_handle_t embedding_handle;
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_malloc(&embedding_handle,
                                   params.get_embedding_table_size(),
                                   wm_comm,
                                   params.get_memory_type(),
                                   params.get_memory_location(),
                                   embedding_entry_size) == WHOLEMEMORY_SUCCESS);

      hipStream_t stream;
      WM_CUDA_CHECK_NO_THROW(hipStreamCreate(&stream));

      void *dev_indices = nullptr, *dev_gather_buffer = nullptr;
      void *host_indices = nullptr;
      size_t gather_buffer_size  = params.get_gather_size();
      size_t indices_buffer_size = wholememory_get_memory_size_from_array(&indices_desc);

      WM_CUDA_CHECK_NO_THROW(hipHostMalloc(&host_indices, indices_buffer_size));
      WM_CUDA_CHECK_NO_THROW(hipMalloc(&dev_indices, indices_buffer_size));
      WM_CUDA_CHECK_NO_THROW(hipMalloc(&dev_gather_buffer, gather_buffer_size));

      wholegraph::bench::host_random_init_integer_indices(
        host_indices, indices_desc, embedding_desc.sizes[0]);
      WM_CUDA_CHECK_NO_THROW(hipMemcpyAsync(dev_indices,
                                host_indices,
                                wholememory_get_memory_size_from_array(&indices_desc),
                                hipMemcpyHostToDevice,
                                stream));
      WM_CUDA_CHECK_NO_THROW(hipStreamSynchronize(stream));
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_communicator_barrier(wm_comm) == WHOLEMEMORY_SUCCESS);

      wholememory_tensor_t embedding_tensor;
      wholememory_tensor_description_t embedding_tensor_desc;
      wholememory_copy_matrix_desc_to_tensor(&embedding_tensor_desc, &embedding_desc);
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_make_tensor_from_handle(
                  &embedding_tensor, embedding_handle, &embedding_tensor_desc) == WHOLEMEMORY_SUCCESS);

      wholememory_tensor_t indices_tensor, output_tensor;
      wholememory_tensor_description_t indices_tensor_desc, output_tensor_desc;
      wholememory_copy_array_desc_to_tensor(&indices_tensor_desc, &indices_desc);
      wholememory_copy_matrix_desc_to_tensor(&output_tensor_desc, &output_desc);
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_make_tensor_from_pointer(&indices_tensor, dev_indices, &indices_tensor_desc) == WHOLEMEMORY_SUCCESS);
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_make_tensor_from_pointer(
                  &output_tensor, dev_gather_buffer, &output_tensor_desc) == WHOLEMEMORY_SUCCESS);
      WM_CUDA_CHECK_NO_THROW(hipStreamSynchronize(stream));
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_communicator_barrier(wm_comm) == WHOLEMEMORY_SUCCESS);

      const auto barrier_fn = [&wm_comm]() -> void { WHOLEMEMORY_CHECK_NOTHROW(wholememory_communicator_barrier(wm_comm) == WHOLEMEMORY_SUCCESS); };

      double emb_size_mb = (double)params.get_embedding_table_size()/1024.0/1024.0;
      double gather_size_mb = (double)params.get_gather_size()/1024.0/1024.0;
      if (world_rank == 0) {
        printf("%s, world_size=%d, memoryType=%s, memoryLocation=%s, elt_size=%ld, embeddingDim=%ld, embeddingTableSize=%.2lf MB, gatherSize=%.2lf MB\n",
            test_type.c_str(), world_size, get_memory_type_string(params.get_memory_type()).c_str(), get_memory_location_string(params.get_memory_location()).c_str(), wholememory_dtype_get_element_size(params.get_embedding_type()), params.get_embedding_dim(), emb_size_mb, gather_size_mb);
      }

      PerformanceMeter meter;
      meter.SetRunCount(100).AddMetrics("Bandwidth", "GB/s",  gather_buffer_size / 1000.0 / 1000.0 / 1000.0, false).SetMaxRunSeconds(1000).SetRunCount(params.get_loop_count());

      if (test_type.compare("gather") == 0) {
        MultiProcessMeasurePerformance([&]{ wholememory_gather(embedding_tensor,
                                                              indices_tensor,
                                                              output_tensor,
                                                              wholememory::get_default_env_func(),
                                                              stream);}, wm_comm, meter, barrier_fn);
        
      }
      else if (test_type.compare("scatter") == 0) {
        MultiProcessMeasurePerformance([&]{
                                      wholememory_scatter(output_tensor,
                                      indices_tensor,
                                      embedding_tensor,
                                      wholememory::get_default_env_func(),
                                      stream);}, wm_comm, meter, barrier_fn);
      }
      else {
        printf("Invalid test function, should be: gather or scatter\n");
        exit(EXIT_FAILURE);
      }

      WHOLEMEMORY_CHECK_NOTHROW(wholememory_destroy_tensor(indices_tensor) == WHOLEMEMORY_SUCCESS);
      WHOLEMEMORY_CHECK_NOTHROW(wholememory_destroy_tensor(output_tensor) == WHOLEMEMORY_SUCCESS);

      WM_CUDA_CHECK_NO_THROW(hipHostFree(host_indices));
      WM_CUDA_CHECK_NO_THROW(hipFree(dev_indices));
      WM_CUDA_CHECK_NO_THROW(hipFree(dev_gather_buffer));

      WHOLEMEMORY_CHECK_NOTHROW(wholememory_destroy_tensor(embedding_tensor) == WHOLEMEMORY_SUCCESS);

      WHOLEMEMORY_CHECK_NOTHROW(wholememory_free(embedding_handle) == WHOLEMEMORY_SUCCESS);

      WHOLEMEMORY_CHECK_NOTHROW(wholememory::destroy_all_communicators() == WHOLEMEMORY_SUCCESS);

      WHOLEMEMORY_CHECK_NOTHROW(wholememory_finalize() == WHOLEMEMORY_SUCCESS);
    },
    true);
}

}  // namespace wholegraph::bench::gather_scatter

int main(int argc, char** argv) {
    wholegraph::bench::gather_scatter::GatherScatterBenchParam params;
    const char* optstr = "ht:l:e:g:d:c:f:";
    struct option opts[] = {
        {"help", no_argument, NULL, 'h'},
        {"memory_type", required_argument, NULL, 't'},        // 0: None, 1: Continuous, 2: Chunked, 3 Distributed
        {"memory_location", required_argument, NULL, 'l'},    // 0: None, 1: Device, 2: Host
        {"embedding_table_size", required_argument, NULL, 'e'},
        {"gather_size", required_argument, NULL, 'g'},
        {"embedding_dim", required_argument, NULL, 'd'},
        {"loop_count", required_argument, NULL, 'c'},
        {"test_type", required_argument, NULL, 'f'}                   //test_type: gather or scatter
    };

    const char *usage = "Usage: %s [options]\n"
                  "Options:\n"
                  "  -h, --help      display this help and exit\n"
                  "  -t, --memory_type   specify wholememory type, 0: None, 1: Continuous, 2: Chunked, 3: Distributed\n"
                  "  -l, --memory_location    specify wholememory location, 0: None, 1: Device, 2: Host\n"
                  "  -e, --embedding_table_size    specify embedding table size\n"
                  "  -g, --gather_size    specify gather size\n"
                  "  -d, --embedding_dim    specify embedding dimension\n"
                  "  -c, --loop_count    specify loop count\n"
                  "  -f, --test_type    specify test type: gather or scatter\n";

    int c;
    bool has_option = false;
    while((c = getopt_long(argc, argv, optstr, opts, NULL)) != -1) {
        has_option = true;
        switch (c)
        {
        char *endptr;
        long val;
        case 'h':
            printf(usage, argv[0]);
            exit(EXIT_SUCCESS);
        case 't':
            val = strtol(optarg,&endptr, 10);
            if (*endptr != '\0' || val < 0 || val > 3) {
                printf("Invalid argument for option -t\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
              params.set_memory_type(static_cast<wholememory_memory_type_t>(val));
            break;
        case 'l':
            val = strtol(optarg,&endptr, 10);
            if (*endptr != '\0' || val < 0 || val > 2) {
                printf("Invalid argument for option -l\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            params.set_memory_location(static_cast<wholememory_memory_location_t>(val));
            break;
        case 'e':
            val = std::stoll(optarg);
            if (val < 0) {
              printf("Negative value, invalid argument for option -e\n");
              printf(usage, argv[0]);
              exit(EXIT_FAILURE);
            }
            params.set_embedding_table_size(val);
            break;
        case 'g':
            val = std::stoll(optarg);
            if (val < 0) {
              printf("Negative value, invalid argument for option -g\n");
              printf(usage, argv[0]);
              exit(EXIT_FAILURE);
            }
            params.set_gather_size(val);
            break;
        case 'd':
                val = std::stoll(optarg);
                if (val < 0) {
                  printf("Negative value, invalid argument for option -d\n");
                  printf(usage, argv[0]);
                  exit(EXIT_FAILURE);
                }
                params.set_embedding_dim(val);
            break;
        case 'c':
            val = std::stoi(optarg);
            if (val < 0) {
              printf("Negative value, invalid argument for option -c\n");
              printf(usage, argv[0]);
              exit(EXIT_FAILURE);
            }
            params.set_loop_count(val);
            break;
        case 'f':
            if (strcmp(optarg, "gather") == 0) {
                params.set_test_type("gather");
            }
            else if (strcmp(optarg, "scatter") == 0) {
                params.set_test_type("scatter");
            }
            else {
                printf("Invalid argument for option -f\n");
                printf(usage, argv[0]);
                exit(EXIT_FAILURE);
            }
            break;
        default:
            printf("Invalid or unrecognized option\n");
            printf(usage, argv[0]);
            exit(EXIT_FAILURE);
        }
    }
    if (!has_option) {
      printf("No option or argument is passed, use the default param\n");
    }
    wholegraph::bench::gather_scatter::gather_scatter_benchmark(params);
    return 0;
}